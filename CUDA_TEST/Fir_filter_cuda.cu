#include "hip/hip_runtime.h"
#define __cplusplus
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "Fir_filter_cuda.cuh"
#include<stdio.h>
#include"hip/device_functions.h"


__global__ void DoFIRFilt(const float* NUM, const int NUMLEN, const float* data, float* out, int datalen)
{
	extern __shared__ float tempsums[];
	const int tid = threadIdx.x;
	int did = blockIdx.x;
	for (size_t didx = did; didx < datalen; didx += gridDim.x)
	{
		float sum = 0;
		int i = tid;
		while (i < NUMLEN)
		{
			sum += NUM[i] * data[didx + i];
			i += blockDim.x;
		}
		tempsums[tid] = sum;
		__syncthreads();
		int numthreads = blockDim.x;
		//reduce
		if (numthreads >= 1024)
		{
			if (tid < 512)
				tempsums[tid] += tempsums[tid + 512];
			__syncthreads();
		}

		if (numthreads >= 512)
		{
			if (tid < 256)
				tempsums[tid] += tempsums[tid + 256];
			__syncthreads();
		}

		if (numthreads >= 256)
		{
			if (tid < 128)
				tempsums[tid] += tempsums[tid + 128];
			__syncthreads();
		}

		if (numthreads >= 128)
		{
			if (tid < 64)
				tempsums[tid] += tempsums[tid + 64];
			__syncthreads();
		}

		if (tid < 32)
		{
			volatile float* wssum = tempsums;
			if (numthreads >= 64)
				wssum[tid] += wssum[tid + 32];
			if (numthreads >= 32)
				wssum[tid] += wssum[tid + 16];
			if (numthreads >= 16)
				wssum[tid] += wssum[tid + 8];
			if (numthreads >= 8)
				wssum[tid] += wssum[tid + 4];
			if (numthreads >= 4)
				wssum[tid] += wssum[tid + 2];
			if (numthreads >= 2)
				wssum[tid] += wssum[tid + 1];

			if (tid == 0)
				out[didx] = wssum[0];
		}

		/*int j = blockDim.x/2;
		while (j > 0)
		{
			if(tid<j)
				tempsums[tid] += tempsums[tid + j];
			__syncthreads();
			j /= 2;
		}

		if (tid == 0)
			out[didx] = tempsums[0];*/
	}
}

extern "C" void fir_cuda(const float* NUM, int NUMLEN, float* data, int datalen, float* outputdata)
{
	int delay = (NUMLEN - 1) / 2;
	float* inputdata = new float[delay + datalen]();
	int i = 0;
	int inputlen = delay + datalen;
	while (i < datalen)
	{
		inputdata[i] = data[i];
		++i;
	}
	float* tempnum = new float[NUMLEN];
	i = 0;
	while (i < NUMLEN)
	{
		tempnum[i] = NUM[NUMLEN - 1 - i];
		++i;
	}
	hipError_t cudaStatus;
	float* d_in = nullptr;
	float* d_out = nullptr;
	float* d_NUM = nullptr;
	cudaStatus = hipMalloc((void**)& d_in, sizeof(float) * inputlen);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

	}
	cudaStatus = hipMalloc((void**)& d_out, sizeof(float) * datalen);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

	}
	cudaStatus = hipMalloc((void**)& d_NUM, sizeof(float) * NUMLEN);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

	}
	cudaStatus = hipMemcpy(d_in, inputdata, sizeof(float) * inputlen, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "copy failed!  Do you have a CUDA-capable GPU installed?");

	}
	cudaStatus = hipMemcpy(d_NUM, tempnum, sizeof(float) * NUMLEN, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "copy failed!  Do you have a CUDA-capable GPU installed?");

	}

	int threadnum = NUMLEN;
	while ((threadnum & (threadnum - 1)) != 0)
		threadnum &= (threadnum - 1);
	if (threadnum > 2048)
		threadnum = 2048;
	int blocknum = (datalen < 100000 ? datalen : 100000);
	dim3 dimgrid(blocknum);
	dim3 blockdim(threadnum);
	size_t tnum = threadnum * sizeof(float);
	DoFIRFilt << <dimgrid, blockdim, tnum >> > (d_NUM, NUMLEN, d_in, d_out, datalen);


	cudaStatus = hipMemcpy(outputdata, d_out, sizeof(float) * datalen, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

	}
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	hipFree(d_in);
	hipFree(d_out);
	delete[] inputdata;
	delete[] tempnum;
}